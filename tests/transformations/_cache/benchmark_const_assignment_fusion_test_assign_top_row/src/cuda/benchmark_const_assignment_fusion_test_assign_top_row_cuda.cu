#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct benchmark_const_assignment_fusion_test_assign_top_row_state_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, int M, int N);
DACE_EXPORTED int __dace_exit_cuda(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state);



int __dace_init_cuda(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    DACE_GPU_CHECK(hipMalloc((void **) &dev_X, 1));
    DACE_GPU_CHECK(hipFree(dev_X));

    

    __state->gpu_context = new dace::cuda::Context(1, 1);

    // Create cuda streams and events
    for(int i = 0; i < 1; ++i) {
        DACE_GPU_CHECK(hipStreamCreateWithFlags(&__state->gpu_context->internal_streams[i], hipStreamNonBlocking));
        __state->gpu_context->streams[i] = __state->gpu_context->internal_streams[i]; // Allow for externals to modify streams
    }
    for(int i = 0; i < 1; ++i) {
        DACE_GPU_CHECK(hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming));
    }

    

    return 0;
}

int __dace_exit_cuda(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state) {
    

    // Synchronize and check for CUDA errors
    int __err = static_cast<int>(__state->gpu_context->lasterror);
    if (__err == 0)
        __err = static_cast<int>(hipDeviceSynchronize());

    // Destroy cuda streams and events
    for(int i = 0; i < 1; ++i) {
        DACE_GPU_CHECK(hipStreamDestroy(__state->gpu_context->internal_streams[i]));
    }
    for(int i = 0; i < 1; ++i) {
        DACE_GPU_CHECK(hipEventDestroy(__state->gpu_context->events[i]));
    }

    delete __state->gpu_context;
    return __err;
}

DACE_EXPORTED bool __dace_gpu_set_stream(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, int streamid, gpuStream_t stream)
{
    if (streamid < 0 || streamid >= 1)
        return false;

    __state->gpu_context->streams[streamid] = stream;

    return true;
}

DACE_EXPORTED void __dace_gpu_set_all_streams(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, gpuStream_t stream)
{
    for (int i = 0; i < 1; ++i)
        __state->gpu_context->streams[i] = stream;
}

__global__ void __launch_bounds__(32) map_fusion_wrapper_0_0_9(float * __restrict__ gpu_A, int M, int N) {
    {
        int t = (blockIdx.x * 32 + threadIdx.x);
        if (t < (Max((M - 1), (N - 1)) + 1)) {
            {
                for (auto gsl_t = t; gsl_t < N; gsl_t += N) {
                    {
                        float __out;

                        ///////////////////
                        // Tasklet code (assign_19_8)
                        __out = 1;
                        ///////////////////

                        gpu_A[t] = __out;
                    }
                    {
                        float __out;

                        ///////////////////
                        // Tasklet code (assign_25_8)
                        __out = 1;
                        ///////////////////

                        gpu_A[(t + (N * (M - 1)))] = __out;
                    }
                }
            }
            {
                for (auto gsl_t = t; gsl_t < M; gsl_t += M) {
                    {
                        float __out;

                        ///////////////////
                        // Tasklet code (assign_31_8)
                        __out = 1;
                        ///////////////////

                        gpu_A[(t * N)] = __out;
                    }
                    {
                        float __out;

                        ///////////////////
                        // Tasklet code (assign_37_8)
                        __out = 1;
                        ///////////////////

                        gpu_A[(((t * N) + N) - 1)] = __out;
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_map_fusion_wrapper_0_0_9(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, float * __restrict__ gpu_A, int M, int N);
void __dace_runkernel_map_fusion_wrapper_0_0_9(benchmark_const_assignment_fusion_test_assign_top_row_state_t *__state, float * __restrict__ gpu_A, int M, int N)
{

    if ((int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32)) == 0) {

        return;
    }

    void  *map_fusion_wrapper_0_0_9_args[] = { (void *)&gpu_A, (void *)&M, (void *)&N };
    gpuError_t __err = hipLaunchKernel((void*)map_fusion_wrapper_0_0_9, dim3(int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32), 1, 1), dim3(32, 1, 1), map_fusion_wrapper_0_0_9_args, 0, __state->gpu_context->streams[0]);
    DACE_KERNEL_LAUNCH_CHECK(__err, "map_fusion_wrapper_0_0_9", int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32), 1, 1, 32, 1, 1);
}

