#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t {
    dace::cuda::Context *gpu_context;
};



DACE_EXPORTED int __dace_init_cuda(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, int K, int M, int N);
DACE_EXPORTED int __dace_exit_cuda(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state);



int __dace_init_cuda(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, int K, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    DACE_GPU_CHECK(hipMalloc((void **) &dev_X, 1));
    DACE_GPU_CHECK(hipFree(dev_X));

    

    __state->gpu_context = new dace::cuda::Context(3, 5);

    // Create cuda streams and events
    for(int i = 0; i < 3; ++i) {
        DACE_GPU_CHECK(hipStreamCreateWithFlags(&__state->gpu_context->internal_streams[i], hipStreamNonBlocking));
        __state->gpu_context->streams[i] = __state->gpu_context->internal_streams[i]; // Allow for externals to modify streams
    }
    for(int i = 0; i < 5; ++i) {
        DACE_GPU_CHECK(hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming));
    }

    

    return 0;
}

int __dace_exit_cuda(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state) {
    

    // Synchronize and check for CUDA errors
    int __err = static_cast<int>(__state->gpu_context->lasterror);
    if (__err == 0)
        __err = static_cast<int>(hipDeviceSynchronize());

    // Destroy cuda streams and events
    for(int i = 0; i < 3; ++i) {
        DACE_GPU_CHECK(hipStreamDestroy(__state->gpu_context->internal_streams[i]));
    }
    for(int i = 0; i < 5; ++i) {
        DACE_GPU_CHECK(hipEventDestroy(__state->gpu_context->events[i]));
    }

    delete __state->gpu_context;
    return __err;
}

DACE_EXPORTED bool __dace_gpu_set_stream(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, int streamid, gpuStream_t stream)
{
    if (streamid < 0 || streamid >= 3)
        return false;

    __state->gpu_context->streams[streamid] = stream;

    return true;
}

DACE_EXPORTED void __dace_gpu_set_all_streams(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, gpuStream_t stream)
{
    for (int i = 0; i < 3; ++i)
        __state->gpu_context->streams[i] = stream;
}

__global__ void __launch_bounds__(32) map_fusion_wrapper_0_0_15(float * __restrict__ gpu_A, int K, int M, int N) {
    {
        {
            int t2 = (blockIdx.x * 32 + threadIdx.x);
            int t1 = (blockIdx.y * 1 + threadIdx.y);
            if (t2 < (Max((M - 1), (N - 1)) + 1)) {
                {
                    {
                        for (auto gsl_t1 = t1; gsl_t1 < (Max((K - 1), (M - 1)) + 1); gsl_t1 += (Max((K - 1), (M - 1)) + 1)) {
                            for (auto gsl_t2 = t2; gsl_t2 < N; gsl_t2 += N) {
                                {
                                    for (auto gsl_t1 = t1; gsl_t1 < M; gsl_t1 += M) {
                                        for (auto gsl_t2 = t2; gsl_t2 < N; gsl_t2 += N) {
                                            {
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (assign_62_8)
                                                __out = 1;
                                                ///////////////////

                                                gpu_A[((N * t1) + t2)] = __out;
                                            }
                                            {
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (assign_68_8)
                                                __out = 1;
                                                ///////////////////

                                                gpu_A[((((M * N) * (K - 1)) + (N * t1)) + t2)] = __out;
                                            }
                                        }
                                    }
                                }
                                {
                                    for (auto gsl_t1 = t1; gsl_t1 < K; gsl_t1 += K) {
                                        for (auto gsl_t2 = t2; gsl_t2 < N; gsl_t2 += N) {
                                            {
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (assign_74_8)
                                                __out = 1;
                                                ///////////////////

                                                gpu_A[(((M * N) * t1) + t2)] = __out;
                                            }
                                            {
                                                float __out;

                                                ///////////////////
                                                // Tasklet code (assign_80_8)
                                                __out = 1;
                                                ///////////////////

                                                gpu_A[((((M * N) * t1) + (N * (M - 1))) + t2)] = __out;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                    {
                        for (auto gsl_t1 = t1; gsl_t1 < K; gsl_t1 += K) {
                            for (auto gsl_t2 = t2; gsl_t2 < M; gsl_t2 += M) {
                                {
                                    float __out;

                                    ///////////////////
                                    // Tasklet code (assign_86_8)
                                    __out = 1;
                                    ///////////////////

                                    gpu_A[(((M * N) * t1) + (N * t2))] = __out;
                                }
                                {
                                    float __out;

                                    ///////////////////
                                    // Tasklet code (assign_92_8)
                                    __out = 1;
                                    ///////////////////

                                    gpu_A[(((((M * N) * t1) + (N * t2)) + N) - 1)] = __out;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_map_fusion_wrapper_0_0_15(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, float * __restrict__ gpu_A, int K, int M, int N);
void __dace_runkernel_map_fusion_wrapper_0_0_15(benchmark_const_assignment_fusion_test_assign_bounary_3d_state_t *__state, float * __restrict__ gpu_A, int K, int M, int N)
{

    if ((int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32)) == 0 || (int_ceil(int_ceil((Max((K - 1), (M - 1)) + 1), 1), 1)) == 0) {

        return;
    }

    void  *map_fusion_wrapper_0_0_15_args[] = { (void *)&gpu_A, (void *)&K, (void *)&M, (void *)&N };
    gpuError_t __err = hipLaunchKernel((void*)map_fusion_wrapper_0_0_15, dim3(int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32), int_ceil(int_ceil((Max((K - 1), (M - 1)) + 1), 1), 1), 1), dim3(32, 1, 1), map_fusion_wrapper_0_0_15_args, 0, __state->gpu_context->streams[0]);
    DACE_KERNEL_LAUNCH_CHECK(__err, "map_fusion_wrapper_0_0_15", int_ceil(int_ceil((Max((M - 1), (N - 1)) + 1), 1), 32), int_ceil(int_ceil((Max((K - 1), (M - 1)) + 1), 1), 1), 1, 32, 1, 1);
}

